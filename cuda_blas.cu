#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <hipsparse.h> 
#include "cuda_blas.h"
#if USE_FP64
#define cuda_data_type HIP_R_64F
#else
#define cuda_data_type HIP_R_32F
#endif
static hipblasHandle_t handle_cublas;
static hipsparseHandle_t handle_cusparse;

static void *mv_buffer;
static void *L_buffer;
static void *U_buffer;
static void *ichol_buffer; // in ichol, we can get away with one buffer

static hipsparseSpMatDescr_t matA = NULL;
static hipsparseSpMatDescr_t matL;
static hipsparseSpMatDescr_t matU;
static hipsparseMatDescr_t descrL, descrU, descrLt, descrM; // last two are used only for incomplete CHolesky
static  csrsv2Info_t infoL, infoU, infoLt;
csric02Info_t infoM  = 0; // used only for Incomplete Cholesky

#define policy HIPSPARSE_SOLVE_POLICY_USE_LEVEL 


void initialize_handles(){
  //printf("initializing handles! \n");
  hipblasCreate(&handle_cublas);
  hipsparseCreate(&handle_cusparse);
}

void initialize_spmv_buffer(const int n, 
                            const int nnz, 
                            int *ia, 
                            int *ja, 
                            real_type *a, 
                            const real_type *x, 
                            real_type *result, 
                            real_type *al, 
                            real_type *bet){
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  size_t mv_buffer_size;
  hipsparseStatus_t status_cusparse;

  status_cusparse = hipsparseCreateDnVec(&vecX,
                                        n,
                                        (void*) x,
                                   cuda_data_type);

  // printf("matX creation status %d\n", status_cusparse);  
  status_cusparse = hipsparseCreateDnVec(&vecY,
                                        n,
                                        (void *) result,
                                         cuda_data_type);

  // printf("vecY creation status %d\n", status_cusparse);  
  status_cusparse = hipsparseCreateCsr(&matA,
                                      n,
                                      n,
                                      nnz,
                                      ia,
                                      ja,
                                      a,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO,
                                       cuda_data_type);

  // printf("matA creation status %d\n", status_cusparse);  
  status_cusparse = hipsparseSpMV_bufferSize(handle_cusparse,
                                            HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                            al,
                                            matA,
                                            vecX,
                                            bet,
                                            vecY,
                                             cuda_data_type,
#if V100
                                             HIPSPARSE_CSRMV_ALG2,
#else
                                             HIPSPARSE_SPMV_CSR_ALG2,
#endif     
                                       &mv_buffer_size);

  hipDeviceSynchronize();

  // printf("mv buffer size %d alpha %f beta %f status %d \n", mv_buffer_size, *al, *bet, status_cusparse);
  hipError_t t = hipMalloc( &mv_buffer, mv_buffer_size);

  if (t != 0) printf("allocated mv_buffer: is it NULL? %d, error %d \n", mv_buffer == NULL, t);

  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
}

void initialize_and_analyze_L_and_U_solve(const int n, 
                                          const int nnzL, 
                                          int *lia, 
                                          int *lja, 
                                          real_type *la,
                                          const int nnzU, 
                                          int *uia, 
                                          int *uja, 
                                          real_type *ua){

  hipsparseCreateMatDescr(&(descrL));
  hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);

  hipsparseCreateMatDescr(&(descrU));
  hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
  hipsparseCreateCsrsv2Info(&infoL);
  hipsparseCreateCsrsv2Info(&infoU);
  int L_buffer_size;  
  int U_buffer_size;  
#if USE_FP64 // it is double 
    hipsparseDcsrsv2_bufferSize(handle_cusparse, 
                               HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               n, 
                               nnzL, 
                               descrL,
                               la, 
                               lia, 
                               lja,
                               infoL, 
                               &L_buffer_size);
    //printf("buffer size L %d\n", L_buffer_size);
    hipMalloc((void**)&(L_buffer), L_buffer_size);

    hipsparseDcsrsv2_bufferSize(handle_cusparse, 
                               HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               n, 
                               nnzU, 
                               descrU,
                               ua, 
                               uia, 
                               uja,
                               infoU, 
                               &U_buffer_size);
    //printf("buffer size U %d\n", U_buffer_size);
    hipMalloc((void**)&(U_buffer), U_buffer_size);
    hipsparseStatus_t status_cusparse;
    status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               n,
                                               nnzL,
                                               descrL,
                                               la,
                                               lia,
                                               lja,
                                               infoL,
                                               policy, 
                                               L_buffer);

    status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               n,
                                               nnzU,
                                               descrU,
                                               ua,
                                               uia,
                                               uja,
                                               infoU,
                                               policy, 
                                               U_buffer);
#else    
// it is 4 bytes so SINGLE

    hipsparseScsrsv2_bufferSize(handle_cusparse, 
                               HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               n, 
                               nnzL, 
                               descrL,
                               la, 
                               lia, 
                               lja,
                               infoL, 
                               &L_buffer_size);
    //printf("buffer size L %d\n", L_buffer_size);
    hipMalloc((void**)&(L_buffer), L_buffer_size);

    hipsparseScsrsv2_bufferSize(handle_cusparse, 
                               HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               n, 
                               nnzU, 
                               descrU,
                               ua, 
                               uia, 
                               uja,
                               infoU, 
                               &U_buffer_size);
    //printf("buffer size U %d\n", U_buffer_size);
    hipMalloc((void**)&(U_buffer), U_buffer_size);
    hipsparseStatus_t status_cusparse;
    status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               n,
                                               nnzL,
                                               descrL,
                                               la,
                                               lia,
                                               lja,
                                               infoL,
                                               policy, 
                                               L_buffer);

    status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
                                               HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                               n,
                                               nnzU,
                                               descrU,
                                               ua,
                                               uia,
                                               uja,
                                               infoU,
                                               policy, 
                                               U_buffer);
#endif
}


void initialize_L_and_U_descriptors(const int n, 
                                    const int nnzL, 
                                    int *lia, 
                                    int *lja, 
                                    real_type *la,
                                    const int nnzU, 
                                    int *uia, 
                                    int *uja, 
                                    real_type *ua){

    hipsparseCreateCsr(&matL,
                      n,
                      n,
                      nnzL,
                      lia,
                      lja,
                      la,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO,
                       cuda_data_type);

    hipsparseCreateCsr(&matU,
                      n,
                      n,
                      nnzU,
                      uia,
                      uja,
                      ua,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_BASE_ZERO,
                       cuda_data_type);
}

void initialize_ichol(const int n, 
                      const int nnzA, 
                      int *ia, 
                      int *ja, 
                      real_type *a)
{

  printf("initializing ICHOL \n");
  hipsparseCreateMatDescr(&descrM);
  hipsparseSetMatIndexBase(descrM, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrM, HIPSPARSE_MATRIX_TYPE_GENERAL);

  hipsparseCreateMatDescr(&descrL);
  hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  hipsparseCreateCsric02Info(&infoM);
  hipsparseCreateCsrsv2Info(&infoL);
  hipsparseCreateCsrsv2Info(&infoLt);
  int structural_zero;
  int numerical_zero; 

  hipsparseStatus_t status_cusparse;

  /* figure out the buffer size */

  int bufferSize, bufferSizeL, bufferSizeLt, bufferSizeM;
#if USE_FP64 // it is double 
      status_cusparse =  hipsparseDcsric02_bufferSize(handle_cusparse, 
                                                     n, 
                                                     nnzA,
                                                     descrM, 
                                                     a,
                                                     ia, 
                                                     ja, 
                                                     infoM, 
                                                     &bufferSizeM);

      status_cusparse =  hipsparseDcsrsv2_bufferSize(handle_cusparse, 
                                                    HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                                    n, 
                                                    nnzA,
                                                    descrL, 
                                                    a, 
                                                    ia, 
                                                    ja, 
                                                    infoL, 
                                                    &bufferSizeL);

      status_cusparse =  hipsparseDcsrsv2_bufferSize(handle_cusparse, 
                                                    HIPSPARSE_OPERATION_TRANSPOSE, 
                                                    n, 
                                                    nnzA,
                                                    descrL, 
                                                    a, 
                                                    ia, 
                                                    ja, 
                                                    infoLt, 
                                                    &bufferSizeLt);
#else

      status_cusparse =  hipsparseScsric02_bufferSize(handle_cusparse, 
                                                     n, 
                                                     nnzA,
                                                     descrM, 
                                                     a,
                                                     ia, 
                                                     ja, 
                                                     infoM, 
                                                     &bufferSizeM);

      status_cusparse =  hipsparseScsrsv2_bufferSize(handle_cusparse, 
                                                    HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                                    n, 
                                                    nnzA,
                                                    descrL, 
                                                    a, 
                                                    ia, 
                                                    ja, 
                                                    infoL, 
                                                    &bufferSizeL);

      status_cusparse =  hipsparseScsrsv2_bufferSize(handle_cusparse, 
                                                    HIPSPARSE_OPERATION_TRANSPOSE, 
                                                    n, 
                                                    nnzA,
                                                    descrL, 
                                                    a, 
                                                    ia, 
                                                    ja, 
                                                    infoLt, 
                                                    &bufferSizeLt);
#endif

  bufferSize = max(bufferSizeM, max(bufferSizeL, bufferSizeLt));

  hipMalloc((void**) &ichol_buffer, bufferSize);

  /* and now analyze */

#if USE_FP64 // it is double 
  status_cusparse = hipsparseDcsric02_analysis(handle_cusparse,
                                              n, 
                                              nnzA, 
                                              descrM,
                                              a, 
                                              ia, 
                                              ja, 
                                              infoM,
                                              policy, 
                                              ichol_buffer);
  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, infoM, &structural_zero);

  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have a problem: A(%d,%d) is missing\n", structural_zero, structural_zero);
  }

  /* analyze the solves as well */

  status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
                                             HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                             n, 
                                             nnzA, 
                                             descrL,
                                             a, 
                                             ia, 
                                             ja,
                                             infoL, 
                                             policy, 
                                             ichol_buffer);

  status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
                                             HIPSPARSE_OPERATION_TRANSPOSE, 
                                             n, 
                                             nnzA, 
                                             descrL,
                                             a, 
                                             ia, 
                                             ja,
                                             infoLt, 
                                             policy, 
                                             ichol_buffer);

  /* decompose */
  status_cusparse = hipsparseDcsric02(handle_cusparse, 
                                     n, 
                                     nnzA, 
                                     descrM,
                                     a, 
                                     ia, 
                                     ja, 
                                     infoM, 
                                     policy, 
                                     ichol_buffer);

  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, 
                                               infoM, 
                                               &numerical_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have another problem: L(%d,%d) is zero\n", numerical_zero, numerical_zero);
  }
#else

  status_cusparse = hipsparseScsric02_analysis(handle_cusparse,
                                              n, 
                                              nnzA, 
                                              descrM,
                                              a, 
                                              ia, 
                                              ja, 
                                              infoM,
                                              policy, 
                                              ichol_buffer);
  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, infoM, &structural_zero);

  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have a problem: A(%d,%d) is missing\n", structural_zero, structural_zero);
  }

  /* analyze the solves as well */

  status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
                                             HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                             n, 
                                             nnzA, 
                                             descrL,
                                             a, 
                                             ia, 
                                             ja,
                                             infoL, 
                                             policy, 
                                             ichol_buffer);

  status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
                                             HIPSPARSE_OPERATION_TRANSPOSE, 
                                             n, 
                                             nnzA, 
                                             descrL,
                                             a, 
                                             ia, 
                                             ja,
                                             infoLt, 
                                             policy, 
                                             ichol_buffer);

  /* decompose */
  status_cusparse = hipsparseScsric02(handle_cusparse, 
                                     n, 
                                     nnzA, 
                                     descrM,
                                     a, 
                                     ia, 
                                     ja, 
                                     infoM, 
                                     policy, 
                                     ichol_buffer);

  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, 
                                               infoM, 
                                               &numerical_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have another problem: L(%d,%d) is zero\n", numerical_zero, numerical_zero);
  }
#endif
}


void cuda_ichol(const int *ia, 
                const int *ja, 
                real_type *a, 
                const int nnzA,
                pdata *prec_data, 
                real_type *x, 
                real_type *y) {
  real_type one = 1.0;

#if USE_FP64 // it is double 
  hipsparseDcsrsv2_solve(handle_cusparse, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        prec_data->n, 
                        nnzA, 
                        &one, 
                        descrL, // replace with cusparseSpSV
                        prec_data->ichol_vals, 
                        ia, 
                        ja, 
                        infoL,
                        x,//input 
                        prec_data->aux_vec1, //output
                        policy, 
                        ichol_buffer);

  /* solve L'*y = aux_vec1 */
  hipsparseDcsrsv2_solve(handle_cusparse, 
                        HIPSPARSE_OPERATION_TRANSPOSE, 
                        prec_data->n, 
                        nnzA, &one, 
                        descrL, // replace with cusparseSpSV
                        prec_data->ichol_vals, 
                        ia, 
                        ja, 
                        infoLt,
                        prec_data->aux_vec1, 
                        y, 
                        policy, 
                        ichol_buffer);
#else //sp

  hipsparseScsrsv2_solve(handle_cusparse, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        prec_data->n, 
                        nnzA, 
                        &one, 
                        descrL, // replace with cusparseSpSV
                        prec_data->ichol_vals, 
                        ia, 
                        ja, 
                        infoL,
                        x,//input 
                        prec_data->aux_vec1, //output
                        policy, 
                        ichol_buffer);

  /* solve L'*y = aux_vec1 */
  hipsparseScsrsv2_solve(handle_cusparse, 
                        HIPSPARSE_OPERATION_TRANSPOSE, 
                        prec_data->n, 
                        nnzA, &one, 
                        descrL, // replace with cusparseSpSV
                        prec_data->ichol_vals, 
                        ia, 
                        ja, 
                        infoLt,
                        prec_data->aux_vec1, 
                        y, 
                        policy, 
                        ichol_buffer);
#endif
}

__global__ void cuda_vec_vec_kernel(const int n,
                                    const real_type *x,
                                    const real_type *y,
                                    real_type *z){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    z[idx] =  x[idx]*y[idx];
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void cuda_vec_reciprocal_kernel(const int n,
                                           const real_type *x,
                                           real_type *z){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    if  (x[idx] != 0.0 ){
      z[idx] = 1.0/x[idx];
    } else {
      z[idx] = 0.0;
    }

    idx += blockDim.x * gridDim.x;
  }
}

__global__ void cuda_vec_sqrt_kernel(const int n,
                                     const real_type *x,
                                     real_type *z){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    if (x[idx]>0) {
      z[idx] =  sqrt(x[idx]);
    } else {
      z[idx] = 0.0;
    }

    idx += blockDim.x * gridDim.x;
  }
}


__global__ void cuda_vec_zero_kernel(const int n,
                                     real_type *x){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    x[idx] =  0.0;

    idx += blockDim.x * gridDim.x;
  }
}

real_type cuda_dot (const int n, const real_type *v, const real_type *w){
  real_type sum;

  hipblasStatus_t status;
#if USE_FP64
  status = hipblasDdot (handle_cublas, 
                       n, 
                       v, 
                       1, 
                       w, 
                       1, 
                       &sum);
#else
  status = hipblasSdot (handle_cublas, 
                       n, 
                       v, 
                       1, 
                       w, 
                       1, 
                       &sum);
#endif
  //printf("DOT product status %d\n", status);
  return sum;
}

void cuda_scal (const int n, const real_type alpha, real_type *v){
#if USE_FP64 
 hipblasDscal(handle_cublas, 
              n,
              &alpha,
              v, 
              1);
#else
 hipblasSscal(handle_cublas, 
              n,
              &alpha,
              v, 
              1);
#endif
}

void cuda_axpy (const int n, const real_type alpha, const real_type *x, real_type *y){

  hipblasStatus_t status;
#if USE_FP64 
 status = hipblasDaxpy(handle_cublas, 
                       n,
                       &alpha,
                       x, 
                       1,
                       y, 
                       1);
#else

 status = hipblasSaxpy(handle_cublas, 
                       n,
                       &alpha,
                       x, 
                       1,
                       y, 
                       1);
#endif
}

void cuda_csr_matvec(const int n, const int nnz, const int *ia, const int *ja, const real_type *a, const real_type *x, real_type *result, const real_type*al, const real_type *bet){
  /* y = alpha *A* x + beta * y */ 

  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseSpMatDescr_t matCSR;
  hipsparseCreateDnVec(&vecX,
                      n,
                      (void*) x,
                      cuda_data_type);

  hipsparseCreateDnVec(&vecY,
                      n,
                      (void *) result,
                      cuda_data_type);

  hipsparseStatus_t status_cusparse;

  status_cusparse = hipsparseCreateCsr(&matCSR,
                                      n,
                                      n,
                                      nnz,
                                      (void *)ia,
                                      (void *)ja,
                                      (void *)a,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO,
                                      cuda_data_type);
  // printf("before matvec: input^Tinput %5.16e, output^Toutput %5.16e alpha %f beta %f\n", cuda_dot(n, x,x), cuda_dot(n, result, result), *al, *bet);
  status_cusparse = hipsparseSpMV(handle_cusparse,
                                 HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 al,
                                 matCSR,
                                 vecX,
                                 bet,
                                 vecY,
                                 cuda_data_type,
#if V100
                                 HIPSPARSE_CSRMV_ALG2,
#else
                                 HIPSPARSE_SPMV_CSR_ALG2,
#endif     
                            mv_buffer);
  //  printf("matvec status: %d is MV BUFFER NULL? %d  is matA null? %d\n", status_cusparse, mv_buffer == NULL, matA==NULL);
  //  printf("after matvec: input^Tinput %5.16e, output^Toutput %5.16e\n", cuda_dot(n, x,x), cuda_dot(n,result, result));

  hipsparseDestroySpMat(matCSR);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
}

void cuda_lower_triangular_solve(const int n,
                                 const int nnzL, 
                                 const int *lia, 
                                 const int *lja, 
                                 const real_type *la,
                                 const real_type *diagonal, 
                                 const real_type *x, real_type *result){
  /* compute result = L^{-1}x */
  /* we DO NOT assume anything about L diagonal */
  /* d_x3 = L^(-1)dx2 */

  real_type one = 1.0;
#if USE_FP64
  hipsparseStatus_t status = hipsparseDcsrsv2_solve(handle_cusparse, 
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                                  n, 
                                                  nnzL, 
                                                  &one, 
                                                  descrL,
                                                  la,
                                                  lia,
                                                  lja,
                                                  infoL,
                                                  x,
                                                  result,
                                                  policy,
                                                  L_buffer);
  //printf("status after tri solve is %d \n", status);
#else
  hipsparseStatus_t status = hipsparseScsrsv2_solve(handle_cusparse, 
                                                  HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                                                  n, 
                                                  nnzL, 
                                                  &one, 
                                                  descrL,
                                                  la,
                                                  lia,
                                                  lja,
                                                  infoL,
                                                  x,
                                                  result,
                                                  policy,
                                                  L_buffer);
#endif
}


void cuda_upper_triangular_solve(const int n, 
                                 const int nnzU, 
                                 const int *uia, 
                                 const int *uja, 
                                 const real_type *ua, 
                                 const real_type *diagonal, 
                                 const real_type *x, 
                                 real_type *result){

  /* compute result = U^{-1}x */
  real_type one = 1.0;
#if USE_FP64 
 hipsparseDcsrsv2_solve(handle_cusparse, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        n, 
                        nnzU, 
                        &one, 
                        descrU,
                        ua,
                        uia,
                        uja,
                        infoU,
                        x,
                        result,
                        policy,
                        U_buffer);
#else
 hipsparseScsrsv2_solve(handle_cusparse, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        n, 
                        nnzU, 
                        &one, 
                        descrU,
                        ua,
                        uia,
                        uja,
                        infoU,
                        x,
                        result,
                        policy,
                        U_buffer);
#endif
}

/* not std blas but needed and embarassingly parallel */

/* cuda vec-vec computes an element-wise product (needed for scaling) */

void cuda_vec_vec(const int n, const real_type *x, const real_type *y, real_type *res){

  cuda_vec_vec_kernel<<<1024, 1024>>>(n, x, y, res);
}

/* vector reciprocal computes 1./d */ 

void cuda_vector_reciprocal(const int n, const real_type *v, real_type *res){

  cuda_vec_reciprocal_kernel<<<1024, 1024>>>(n, v, res);
}

/* vector sqrt takes an sqrt from each vector entry */

void cuda_vector_sqrt(const int n, const real_type *v, real_type *res){

  cuda_vec_sqrt_kernel<<<1024, 1024>>>(n, v, res);
}

void cuda_vec_copy(const int n, const real_type *src, real_type *dest){

  hipMemcpy(dest, src, sizeof(real_type) * n, hipMemcpyDeviceToDevice);
}

void cuda_vec_zero(const int n, real_type *vec){

  cuda_vec_zero_kernel<<<1024, 1024>>>(n, vec);
}

