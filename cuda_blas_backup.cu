#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <hipsparse.h> 
#include "cuda_blas.h"
#if USE_FP64
#define cuda_data_type HIP_R_64F
#else
#define cuda_data_type HIP_R_32F
#endif
static hipblasHandle_t handle_cublas;
static hipsparseHandle_t handle_cusparse;

static void *mv_buffer;
static void *L_buffer;
static void *U_buffer;
static void *ichol_buffer; // in ichol, we can get away with one buffer (?)
static void *ichol_Lbuffer;
static void *ichol_Ltbuffer;


static hipsparseSpMatDescr_t matA = NULL;
static hipsparseSpMatDescr_t matL;
static hipsparseSpMatDescr_t matU;
static hipsparseMatDescr_t descrLt, descrM; // last two are used only for incomplete CHolesky
#if (H100 != 1)
static  csrsv2Info_t infoL, infoU, infoLt;
static hipsparseMatDescr_t descrL, descrU;
#else
static hipsparseSpSVDescr_t  descrL, descrU;
static hipsparseSpSVDescr_t descrLichol;
static hipsparseSpSVDescr_t descrLtichol;
#endif
csric02Info_t infoM  = 0; // used only for Incomplete Cholesky

#define policy HIPSPARSE_SOLVE_POLICY_USE_LEVEL 


void initialize_handles(){
  //printf("initializing handles! \n");
  hipblasCreate(&handle_cublas);
  hipsparseCreate(&handle_cusparse);
}

void initialize_spmv_buffer(const int n, 
			    const int nnz, 
			    int *ia, 
			    int *ja, 
			    real_type *a, 
			    const real_type *x, 
			    real_type *result, 
			    real_type *al, 
			    real_type *bet){
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  size_t mv_buffer_size;
  hipsparseStatus_t status_cusparse;

  status_cusparse = hipsparseCreateDnVec(&vecX,
					n,
					(void*) x,
					cuda_data_type);

  // printf("matX creation status %d\n", status_cusparse);  
  status_cusparse = hipsparseCreateDnVec(&vecY,
					n,
					(void *) result,
					cuda_data_type);

  // printf("vecY creation status %d\n", status_cusparse);  
  status_cusparse = hipsparseCreateCsr(&matA,
				      n,
				      n,
				      nnz,
				      ia,
				      ja,
				      a,
				      HIPSPARSE_INDEX_32I,
				      HIPSPARSE_INDEX_32I,
				      HIPSPARSE_INDEX_BASE_ZERO,
				      cuda_data_type);

  // printf("matA creation status %d\n", status_cusparse);  
  status_cusparse = hipsparseSpMV_bufferSize(handle_cusparse,
					    HIPSPARSE_OPERATION_NON_TRANSPOSE,
					    al,
					    matA,
					    vecX,
					    bet,
					    vecY,
					    cuda_data_type,
#if V100
					    HIPSPARSE_CSRMV_ALG2,
#else
					    HIPSPARSE_SPMV_CSR_ALG2,
#endif     
					    &mv_buffer_size);

  hipDeviceSynchronize();

  // printf("mv buffer size %d alpha %f beta %f status %d \n", mv_buffer_size, *al, *bet, status_cusparse);
  hipError_t t = hipMalloc( &mv_buffer, mv_buffer_size);

  if (t != 0) printf("allocated mv_buffer: is it NULL? %d, error %d \n", mv_buffer == NULL, t);

  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
}

void initialize_and_analyze_L_and_U_solve(const int n, 
					  const int nnzL, 
					  int *lia, 
					  int *lja, 
					  real_type *la,
					  const int nnzU, 
					  int *uia, 
					  int *uja, 
					  real_type *ua,
					  real_type *vecx_data,
					  real_type *vecy_data)
{

#if (H100 != 1)
  int L_buffer_size;  
  int U_buffer_size; 
  hipsparseCreateMatDescr(&(descrL));
  hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);

  hipsparseCreateMatDescr(&(descrU));
  hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
  hipsparseCreateCsrsv2Info(&infoL);
  hipsparseCreateCsrsv2Info(&infoU);
#if USE_FP64 // it is double 
  hipsparseDcsrsv2_bufferSize(handle_cusparse, 
			     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			     n, 
			     nnzL, 
			     descrL,
			     la, 
			     lia, 
			     lja,
			     infoL, 
			     &L_buffer_size);
  //printf("buffer size L %d\n", L_buffer_size);
  hipMalloc((void**)&(L_buffer), L_buffer_size);

  hipsparseDcsrsv2_bufferSize(handle_cusparse, 
			     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			     n, 
			     nnzU, 
			     descrU,
			     ua, 
			     uia, 
			     uja,
			     infoU, 
			     &U_buffer_size);
  //printf("buffer size U %d\n", U_buffer_size);
  hipMalloc((void**)&(U_buffer), U_buffer_size);
  hipsparseStatus_t status_cusparse;
  status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE,
					     n,
					     nnzL,
					     descrL,
					     la,
					     lia,
					     lja,
					     infoL,
					     policy, 
					     L_buffer);

  status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE,
					     n,
					     nnzU,
					     descrU,
					     ua,
					     uia,
					     uja,
					     infoU,
					     policy, 
					     U_buffer);
#else    
  // it is 4 bytes so SINGLE

  hipsparseScsrsv2_bufferSize(handle_cusparse, 
			     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			     n, 
			     nnzL, 
			     descrL,
			     la, 
			     lia, 
			     lja,
			     infoL, 
			     &L_buffer_size);
  //printf("buffer size L %d\n", L_buffer_size);
  hipMalloc((void**)&(L_buffer), L_buffer_size);

  hipsparseScsrsv2_bufferSize(handle_cusparse, 
			     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			     n, 
			     nnzU, 
			     descrU,
			     ua, 
			     uia, 
			     uja,
			     infoU, 
			     &U_buffer_size);
  //printf("buffer size U %d\n", U_buffer_size);
  hipMalloc((void**)&(U_buffer), U_buffer_size);
  hipsparseStatus_t status_cusparse;
  status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE,
					     n,
					     nnzL,
					     descrL,
					     la,
					     lia,
					     lja,
					     infoL,
					     policy, 
					     L_buffer);

  status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE,
					     n,
					     nnzU,
					     descrU,
					     ua,
					     uia,
					     uja,
					     infoU,
					     policy, 
					     U_buffer);
#endif
  //end of FP32 code
#else //We ARE using cuda/12 or H100

  size_t L_buffer_size;  
  size_t U_buffer_size; 
#if USE_FP64 // it is double 
  hipsparseCreateCsr(&matL, 
		    n, 
		    n, 
		    nnzL,
		    lia, 
		    lja, 
		    la,
		    HIPSPARSE_INDEX_32I, 
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_BASE_ZERO, 
		    HIP_R_64F);
  hipsparseCreateCsr(&matU, 
		    n, 
		    n, 
		    nnzU,
		    uia, 
		    uja, 
		    ua,
		    HIPSPARSE_INDEX_32I, 
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_BASE_ZERO, 
		    HIP_R_64F);
  //descriptors
  hipsparseSpSV_createDescr(&descrL);
  hipsparseSpSV_createDescr(&descrU);

  //set fill modes
  hipsparseFillMode_t fillmodeL = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t fillmodeU = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_FILL_MODE,
			    &fillmodeL, 
			    sizeof(fillmodeL)); 
  hipsparseSpMatSetAttribute(matU, 
			    HIPSPARSE_SPMAT_FILL_MODE,
			    &fillmodeU, 
			    sizeof(fillmodeU)); 

  //set diag types (note: both are NON UNIT)
  hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_DIAG_TYPE,
			    &diagtype, 
			    sizeof(diagtype));
  hipsparseSpMatSetAttribute(matU, 
			    HIPSPARSE_SPMAT_DIAG_TYPE,
			    &diagtype, 
			    sizeof(diagtype));
  // vectors 
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;

  hipsparseCreateDnVec(&vecX, n, vecx_data, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, n, vecy_data, HIP_R_64F);

  // allocate buffers
  real_type alpha = 1.0;
  hipsparseSpSV_bufferSize(handle_cusparse, 
			  HIPSPARSE_OPERATION_NON_TRANSPOSE,
			  &alpha,
			  matL, 
			  vecX, 
			  vecY, 
			  HIP_R_64F,
			  HIPSPARSE_SPSV_ALG_DEFAULT, 
			  descrL,
			  &L_buffer_size);
  hipMalloc(&L_buffer, L_buffer_size);
  hipsparseSpSV_bufferSize(handle_cusparse, 
			  HIPSPARSE_OPERATION_NON_TRANSPOSE,
			  &alpha,
			  matU, 
			  vecX, 
			  vecY, 
			  HIP_R_64F,
			  HIPSPARSE_SPSV_ALG_DEFAULT, 
			  descrU,
			  &U_buffer_size);
  hipMalloc(&U_buffer, L_buffer_size);
  // analyze
  hipsparseSpSV_analysis(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha,
			matL, 
			vecX, 
			vecY, 
			HIP_R_64F,
			HIPSPARSE_SPSV_ALG_DEFAULT, 
			descrL,
			L_buffer);

  hipsparseSpSV_analysis(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha,
			matU, 
			vecX, 
			vecY, 
			HIP_R_64F,
			HIPSPARSE_SPSV_ALG_DEFAULT, 
			descrU,
			U_buffer);

#else // same code but for single

  hipsparseCreateCsr(&matL, 
		    n, 
		    n, 
		    nnzL,
		    lia, 
		    lja, 
		    la,
		    HIPSPARSE_INDEX_32I, 
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_BASE_ZERO, 
		    HIP_R_32F);
  hipsparseCreateCsr(&matU, 
		    n, 
		    n, 
		    nnzU,
		    uia, 
		    uja, 
		    ua,
		    HIPSPARSE_INDEX_32I, 
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_BASE_ZERO, 
		    HIP_R_32F);
  //descriptors
  hipsparseSpSV_createDescr(&descrL);
  hipsparseSpSV_createDescr(&descrU);

  //set fill modes
  hipsparseFillMode_t fillmodeL = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseFillMode_t fillmodeU = HIPSPARSE_FILL_MODE_UPPER;
  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_FILL_MODE,
			    &fillmodeL, 
			    sizeof(fillmodeL)); 
  hipsparseSpMatSetAttribute(matU, 
			    HIPSPARSE_SPMAT_FILL_MODE,
			    &fillmodeU, 
			    sizeof(fillmodeU)); 

  //set diag types (note: both are NON UNIT)
  hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_DIAG_TYPE,
			    &diagtype, 
			    sizeof(diagtype));
  hipsparseSpMatSetAttribute(matU, 
			    HIPSPARSE_SPMAT_DIAG_TYPE,
			    &diagtype, 
			    sizeof(diagtype));
  // vectors 
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;

  hipsparseCreateDnVec(&vecX, n, vecx_data, HIP_R_32F);
  hipsparseCreateDnVec(&vecY, n, vecy_data, HIP_R_32F);

  // allocate buffers
  real_type alpha = 1.0;
  hipsparseSpSV_bufferSize(handle_cusparse, 
			  HIPSPARSE_OPERATION_NON_TRANSPOSE,
			  &alpha,
			  matL, 
			  vecX, 
			  vecY, 
			  HIP_R_32F,
			  HIPSPARSE_SPSV_ALG_DEFAULT, 
			  descrL,
			  &L_buffer_size);
  hipMalloc(&L_buffer, L_buffer_size);
  hipsparseSpSV_bufferSize(handle_cusparse, 
			  HIPSPARSE_OPERATION_NON_TRANSPOSE,
			  &alpha,
			  matU, 
			  vecX, 
			  vecY, 
			  HIP_R_32F,
			  HIPSPARSE_SPSV_ALG_DEFAULT, 
			  descrU,
			  &U_buffer_size);
  hipMalloc(&U_buffer, L_buffer_size);
  // analyze
  hipsparseSpSV_analysis(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha,
			matL, 
			vecX, 
			vecY, 
			HIP_R_32F,
			HIPSPARSE_SPSV_ALG_DEFAULT, 
			descrL,
			L_buffer);

  hipsparseSpSV_analysis(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			&alpha,
			matU, 
			vecX, 
			vecY, 
			HIP_R_32F,
			HIPSPARSE_SPSV_ALG_DEFAULT, 
			descrU,
			U_buffer);
#endif // F32 vs F64
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
#endif //H100

}


void initialize_L_and_U_descriptors(const int n, 
				    const int nnzL, 
				    int *lia, 
				    int *lja, 
				    real_type *la,
				    const int nnzU, 
				    int *uia, 
				    int *uja, 
				    real_type *ua){

  hipsparseCreateCsr(&matL,
		    n,
		    n,
		    nnzL,
		    lia,
		    lja,
		    la,
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_BASE_ZERO,
		    cuda_data_type);

  hipsparseCreateCsr(&matU,
		    n,
		    n,
		    nnzU,
		    uia,
		    uja,
		    ua,
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_32I,
		    HIPSPARSE_INDEX_BASE_ZERO,
		    cuda_data_type);
}

void initialize_ichol(const int n, 
		      const int nnzA, 
		      int *ia, 
		      int *ja, 
		      real_type *a,
		      real_type *xdata,
		      real_type *ydata)
{
  hipsparseCreateMatDescr(&descrM);
  hipsparseSetMatIndexBase(descrM, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrM, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseCreateCsric02Info(&infoM);

#if (H100 != 1)
  
  hipsparseCreateMatDescr(&descrL);
  hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
  hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_NON_UNIT);

  hipsparseCreateCsrsv2Info(&infoL);
  hipsparseCreateCsrsv2Info(&infoLt);

#endif  

  int structural_zero;
  int numerical_zero; 

  hipsparseStatus_t status_cusparse;

  /* figure out the buffer size */

  int bufferSize, bufferSizeL, bufferSizeLt, bufferSizeM;
#if USE_FP64 // it is double 
  status_cusparse =  hipsparseDcsric02_bufferSize(handle_cusparse, 
						 n, 
						 nnzA,
						 descrM, 
						 a,
						 ia, 
						 ja, 
						 infoM, 
						 &bufferSizeM);

  if (status_cusparse != 0 ) printf("We have a problem (-2), status: %d \n", status_cusparse ); 
#if (H100 != 1)
  
  status_cusparse =  hipsparseDcsrsv2_bufferSize(handle_cusparse, 
						HIPSPARSE_OPERATION_NON_TRANSPOSE, 
						n, 
						nnzA,
						descrL, 
						a, 
						ia, 
						ja, 
						infoL, 
						&bufferSizeL);

  status_cusparse =  hipsparseDcsrsv2_bufferSize(handle_cusparse, 
						HIPSPARSE_OPERATION_TRANSPOSE, 
						n, 
						nnzA,
						descrL, 
						a, 
						ia, 
						ja, 
						infoLt, 
						&bufferSizeLt);
  bufferSize = max(bufferSizeM, max(bufferSizeL, bufferSizeLt));

  hipMalloc((void**) &ichol_buffer, bufferSize);
#else
  // CUDA 12 aka H100
  hipDataType dataType;
  if (sizeof(reaL_type) == 4) dataType = HIP_R_32F;
  if (sizeof(reaL_type) == 8) dataType = HIP_R_64F;
  size_t bL, bLt;
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;

  hipsparseCreateDnVec(&vecX, n, xdata, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, n, ydata, HIP_R_64F);
  double alpha = 1.0;
  status_cusparse = hipsparseCreateCsr(&matL, 
				      n, 
				      n, 
				      nnzA,
				      ia, 
				      ja, 
				      a, 
				      HIPSPARSE_INDEX_32I, 
				      HIPSPARSE_INDEX_32I,
				      HIPSPARSE_INDEX_BASE_ZERO, 
				      dataType);

  if (status_cusparse != 0 ) printf("We have a problem SV (0), status: %d \n", status_cusparse ); 
  
  hipsparseSpSV_createDescr(&descrLichol);
  hipsparseSpSV_createDescr(&descrLtichol);

  //set fill modes
  hipsparseFillMode_t fillmodeL = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_FILL_MODE,
			    &fillmodeL, 
			    sizeof(fillmodeL)); 

  //set diag types (note: both are NON UNIT)
  hipsparseDiagType_t diagtype = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_DIAG_TYPE,
			    &diagtype, 
			    sizeof(diagtype));

  hipsparseSpMatSetAttribute(matL, 
			    HIPSPARSE_SPMAT_DIAG_TYPE,
			    &diagtype, 
			    sizeof(diagtype));

  status_cusparse = hipsparseSpSV_bufferSize(handle_cusparse, 
					    HIPSPARSE_OPERATION_NON_TRANSPOSE,
					    &alpha,
					    matL, 
					    NULL, 
					    NULL, 
					    dataType,
					    HIPSPARSE_SPSV_ALG_DEFAULT, 
					    descrLichol,
					    &bL);
 
 status_cusparse = hipsparseSpSV_bufferSize(handle_cusparse, 
					    HIPSPARSE_OPERATION_TRANSPOSE,
					    &alpha,
					    matL, 
					    NULL, 
					    NULL, 
					    dataType,
					    HIPSPARSE_SPSV_ALG_DEFAULT, 
					    descrLtichol,
					    &bLt);
  if (status_cusparse != 0 ) printf("We have a problem SV (1), status: %d \n", status_cusparse ); 
  bufferSizeL =  (int) bL;
  bufferSizeLt =  (int) bLt;

// we need separate buffers in this case
  hipMalloc((void**) &ichol_buffer, bufferSizeM);
  hipMalloc((void**) &ichol_Lbuffer, bufferSizeL);
  hipMalloc((void**) &ichol_Ltbuffer, bufferSizeLt);
#endif
#else

  status_cusparse =  hipsparseScsric02_bufferSize(handle_cusparse, 
						 n, 
						 nnzA,
						 descrM, 
						 a,
						 ia, 
						 ja, 
						 infoM, 
						 &bufferSizeM);

  status_cusparse =  hipsparseScsrsv2_bufferSize(handle_cusparse, 
						HIPSPARSE_OPERATION_NON_TRANSPOSE, 
						n, 
						nnzA,
						descrLichol, 
						a, 
						ia, 
						ja, 
						infoL, 
						&bufferSizeL);

  status_cusparse =  hipsparseScsrsv2_bufferSize(handle_cusparse, 
						HIPSPARSE_OPERATION_TRANSPOSE, 
						n, 
						nnzA,
						descrLichol, 
						a, 
						ia, 
						ja, 
						infoLt, 
						&bufferSizeLt);
  bufferSize = max(bufferSizeM, max(bufferSizeL, bufferSizeLt));

  hipMalloc((void**) &ichol_buffer, bufferSize);
#endif
  /* and now analyze */

#if USE_FP64 // it is double 
  status_cusparse = hipsparseDcsric02_analysis(handle_cusparse,
					      n, 
					      nnzA, 
					      descrM,
					      a, 
					      ia, 
					      ja, 
					      infoM,
					      policy, 
					      ichol_buffer);
  if (status_cusparse != 0 ) printf("We have a problem (-1), status: %d \n", status_cusparse ); 
  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, infoM, &structural_zero);

  hipDeviceSynchronize();
  /* decompose */
  status_cusparse = hipsparseDcsric02(handle_cusparse, 
				     n, 
				     nnzA, 
				     descrM,
				     a, 
				     ia, 
				     ja, 
				     infoM, 
				     policy, 
				     ichol_buffer);

  hipDeviceSynchronize();
  if (status_cusparse != 0 ) printf("We have a problem (2), status: %d \n", status_cusparse ); 

  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have a problem: A(%d,%d) is missing\n", structural_zero, structural_zero);
  }



  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, 
					       infoM, 
					       &numerical_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have another problem: L(%d,%d) is zero\n", numerical_zero, numerical_zero);
  }
  /* analyze the solves as well */

#if (H100 != 1)
  status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
					     n, 
					     nnzA, 
					     descrLichol,
					     a, 
					     ia, 
					     ja,
					     infoL, 
					     policy, 
					     ichol_buffer);

  status_cusparse = hipsparseDcsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_TRANSPOSE, 
					     n, 
					     nnzA, 
					     descrLichol,
					     a, 
					     ia, 
					     ja,
					     infoLt, 
					     policy, 
					     ichol_buffer);

#else
  // H100 + CUDA 12
  cusparseSpSV_updateMatrix(handle_cusparse,
			    descrLichol,
			    a,
			    CUSPARSE_SPSV_UPDATE_GENERAL);
  status_cusparse = hipsparseSpSV_analysis(handle_cusparse, 
					  HIPSPARSE_OPERATION_NON_TRANSPOSE,
					  &alpha,
					  matL, 
					  NULL, 
					  NULL, 
					  HIP_R_64F,
					  HIPSPARSE_SPSV_ALG_DEFAULT, 
					  descrLichol,
					  ichol_Lbuffer);
  if (status_cusparse != 0 ) printf("We have a problem SV (0), status: %d \n", status_cusparse ); 
  status_cusparse = hipsparseSpSV_analysis(handle_cusparse, 
					  HIPSPARSE_OPERATION_TRANSPOSE,
					  &alpha,
					  matL, 
					  NULL, 
					  NULL, 
					  HIP_R_64F,
					  HIPSPARSE_SPSV_ALG_DEFAULT, 
					  descrLtichol,
					  ichol_Ltbuffer);
  if (status_cusparse != 0 ) printf("We have a problem SV (1), status: %d \n", status_cusparse ); 
#endif
#else

  status_cusparse = hipsparseScsric02_analysis(handle_cusparse,
					      n, 
					      nnzA, 
					      descrM,
					      a, 
					      ia, 
					      ja, 
					      infoM,
					      policy, 
					      ichol_buffer);
  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, infoM, &structural_zero);

  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have a problem: A(%d,%d) is missing\n", structural_zero, structural_zero);
  }

  /* analyze the solves as well */

  status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_NON_TRANSPOSE, 
					     n, 
					     nnzA, 
					     descrLichol,
					     a, 
					     ia, 
					     ja,
					     infoL, 
					     policy, 
					     ichol_buffer);

  status_cusparse = hipsparseScsrsv2_analysis(handle_cusparse, 
					     HIPSPARSE_OPERATION_TRANSPOSE, 
					     n, 
					     nnzA, 
					     descrLichol,
					     a, 
					     ia, 
					     ja,
					     infoLt, 
					     policy, 
					     ichol_buffer);

  /* decompose */
  status_cusparse = hipsparseScsric02(handle_cusparse, 
				     n, 
				     nnzA, 
				     descrM,
				     a, 
				     ia, 
				     ja, 
				     infoM, 
				     policy, 
				     ichol_buffer);

  status_cusparse = hipsparseXcsric02_zeroPivot(handle_cusparse, 
					       infoM, 
					       &numerical_zero);
  if (HIPSPARSE_STATUS_ZERO_PIVOT == status_cusparse) {
    printf("We have another problem: L(%d,%d) is zero\n", numerical_zero, numerical_zero);
  }
#endif
}


void cuda_ichol(const int *ia, 
		const int *ja, 
		real_type *a, 
		const int nnzA,
		pdata *prec_data, 
		real_type *x, 
		real_type *y) {
  real_type one = 1.0;
#if (H100 != 1) 
#if USE_FP64 // it is double
  hipsparseDcsrsv2_solve(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			prec_data->n, 
			nnzA, 
			&one, 
			descrL, // replace with cusparseSpSV
			prec_data->ichol_vals, 
			ia, 
			ja, 
			infoL,
			x,//input 
			prec_data->aux_vec1, //output
			policy, 
			ichol_buffer);

  /* solve L'*y = aux_vec1 */
  hipsparseDcsrsv2_solve(handle_cusparse, 
			HIPSPARSE_OPERATION_TRANSPOSE, 
			prec_data->n, 
			nnzA, &one, 
			descrL, // replace with cusparseSpSV
			prec_data->ichol_vals, 
			ia, 
			ja, 
			infoLt,
			prec_data->aux_vec1, 
			y, 
			policy, 
			ichol_buffer);
#else //sp

  hipsparseScsrsv2_solve(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			prec_data->n, 
			nnzA, 
			&one, 
			descrL, // replace with cusparseSpSV
			prec_data->ichol_vals, 
			ia, 
			ja, 
			infoL,
			x,//input 
			prec_data->aux_vec1, //output
			policy, 
			ichol_buffer);

  /* solve L'*y = aux_vec1 */
  hipsparseScsrsv2_solve(handle_cusparse, 
			HIPSPARSE_OPERATION_TRANSPOSE, 
			prec_data->n, 
			nnzA, &one, 
			descrL, // replace with cusparseSpSV
			prec_data->ichol_vals, 
			ia, 
			ja, 
			infoLt,
			prec_data->aux_vec1, 
			y, 
			policy, 
			ichol_buffer);
#endif
#else //it is H100

  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseDnVecDescr_t vecT;
  hipsparseCreateDnVec(&vecX, prec_data->n,  x, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, prec_data->n,  y, HIP_R_64F);
  hipsparseCreateDnVec(&vecT, prec_data->n,  prec_data->aux_vec1, HIP_R_64F);
  hipsparseSpSV_solve(handle_cusparse, 
		     HIPSPARSE_OPERATION_NON_TRANSPOSE,
		     &one, 
		     matL, 
		     vecX, 
		     vecT, 
		     HIP_R_64F,
		     HIPSPARSE_SPSV_ALG_DEFAULT, 
		     descrLichol); 
  hipsparseSpSV_solve(handle_cusparse, 
		     HIPSPARSE_OPERATION_TRANSPOSE,
		     &one, 
		     matL, 
		     vecT, 
		     vecY, 
		     HIP_R_64F,
		     HIPSPARSE_SPSV_ALG_DEFAULT, 
		     descrLtichol); 
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
  hipsparseDestroyDnVec(vecT);

#endif
}

__global__ void cuda_vec_vec_kernel(const int n,
				    const real_type *x,
				    const real_type *y,
				    real_type *z){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    z[idx] =  x[idx]*y[idx];
    idx += blockDim.x * gridDim.x;
  }
}

__global__ void cuda_vec_reciprocal_kernel(const int n,
					   const real_type *x,
					   real_type *z){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    if  (x[idx] != 0.0 ){
      z[idx] = 1.0/x[idx];
    } else {
      z[idx] = 0.0;
    }

    idx += blockDim.x * gridDim.x;
  }
}

__global__ void cuda_vec_sqrt_kernel(const int n,
				     const real_type *x,
				     real_type *z){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    if (x[idx]>0) {
      z[idx] =  sqrt(x[idx]);
    } else {
      z[idx] = 0.0;
    }

    idx += blockDim.x * gridDim.x;
  }
}


__global__ void cuda_vec_zero_kernel(const int n,
				     real_type *x){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  while (idx < n){
    x[idx] =  0.0;

    idx += blockDim.x * gridDim.x;
  }
}

real_type cuda_dot (const int n, const real_type *v, const real_type *w){
  real_type sum;

  hipblasStatus_t status;
#if USE_FP64
  status = hipblasDdot (handle_cublas, 
		       n, 
		       v, 
		       1, 
		       w, 
		       1, 
		       &sum);
#else
  status = hipblasSdot (handle_cublas, 
		       n, 
		       v, 
		       1, 
		       w, 
		       1, 
		       &sum);
#endif
  //printf("DOT product status %d\n", status);
  return sum;
}

void cuda_scal (const int n, const real_type alpha, real_type *v){
#if USE_FP64 
  hipblasDscal(handle_cublas, 
	      n,
	      &alpha,
	      v, 
	      1);
#else
  hipblasSscal(handle_cublas, 
	      n,
	      &alpha,
	      v, 
	      1);
#endif
}

void cuda_axpy (const int n, const real_type alpha, const real_type *x, real_type *y){

  hipblasStatus_t status;
#if USE_FP64 
  status = hipblasDaxpy(handle_cublas, 
		       n,
		       &alpha,
		       x, 
		       1,
		       y, 
		       1);
#else

  status = hipblasSaxpy(handle_cublas, 
		       n,
		       &alpha,
		       x, 
		       1,
		       y, 
		       1);
#endif
}

void cuda_csr_matvec(const int n, const int nnz, const int *ia, const int *ja, const real_type *a, const real_type *x, real_type *result, const real_type*al, const real_type *bet){
  /* y = alpha *A* x + beta * y */ 

  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseSpMatDescr_t matCSR;
  hipsparseCreateDnVec(&vecX,
		      n,
		      (void*) x,
		      cuda_data_type);

  hipsparseCreateDnVec(&vecY,
		      n,
		      (void *) result,
		      cuda_data_type);

  hipsparseStatus_t status_cusparse;

  status_cusparse = hipsparseCreateCsr(&matCSR,
				      n,
				      n,
				      nnz,
				      (void *)ia,
				      (void *)ja,
				      (void *)a,
				      HIPSPARSE_INDEX_32I,
				      HIPSPARSE_INDEX_32I,
				      HIPSPARSE_INDEX_BASE_ZERO,
				      cuda_data_type);
  // printf("before matvec: input^Tinput %5.16e, output^Toutput %5.16e alpha %f beta %f\n", cuda_dot(n, x,x), cuda_dot(n, result, result), *al, *bet);
  status_cusparse = hipsparseSpMV(handle_cusparse,
				 HIPSPARSE_OPERATION_NON_TRANSPOSE,
				 al,
				 matCSR,
				 vecX,
				 bet,
				 vecY,
				 cuda_data_type,
#if V100
				 HIPSPARSE_CSRMV_ALG2,
#else
				 HIPSPARSE_SPMV_CSR_ALG2,
#endif     
				 mv_buffer);
  //  printf("matvec status: %d is MV BUFFER NULL? %d  is matA null? %d\n", status_cusparse, mv_buffer == NULL, matA==NULL);
  //  printf("after matvec: input^Tinput %5.16e, output^Toutput %5.16e\n", cuda_dot(n, x,x), cuda_dot(n,result, result));

  hipsparseDestroySpMat(matCSR);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
}

void cuda_lower_triangular_solve(const int n,
				 const int nnzL, 
				 const int *lia, 
				 const int *lja, 
				 const real_type *la,
				 const real_type *diagonal, 
				 const real_type *x, 
				 real_type *result){
  /* compute result = L^{-1}x */
  /* we DO NOT assume anything about L diagonal */
  /* d_x3 = L^(-1)dx2 */

  real_type one = 1.0;

#if (H100 != 1)
#if USE_FP64
  hipsparseStatus_t status = hipsparseDcsrsv2_solve(handle_cusparse, 
						  HIPSPARSE_OPERATION_NON_TRANSPOSE, 
						  n, 
						  nnzL, 
						  &one, 
						  descrL,
						  la,
						  lia,
						  lja,
						  infoL,
						  x,
						  result,
						  policy,
						  L_buffer);
  //printf("status after tri solve is %d \n", status);
#else
  hipsparseStatus_t status = hipsparseScsrsv2_solve(handle_cusparse, 
						  HIPSPARSE_OPERATION_NON_TRANSPOSE, 
						  n, 
						  nnzL, 
						  &one, 
						  descrL,
						  la,
						  lia,
						  lja,
						  infoL,
						  x,
						  result,
						  policy,
						  L_buffer);
#endif
#else // cuda 12 or H100
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseCreateDnVec(&vecX, n, (double *) x, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, n, result, HIP_R_64F);
  hipsparseSpSV_solve(handle_cusparse, 
		     HIPSPARSE_OPERATION_NON_TRANSPOSE,
		     &one, 
		     matL, 
		     vecX, 
		     vecY, 
		     HIP_R_64F,
		     HIPSPARSE_SPSV_ALG_DEFAULT, 
		     descrL); 
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);

#endif
}


void cuda_upper_triangular_solve(const int n, 
				 const int nnzU, 
				 const int *uia, 
				 const int *uja, 
				 const real_type *ua, 
				 const real_type *diagonal, 
				 const real_type *x, 
				 real_type *result){

  /* compute result = U^{-1}x */
  real_type one = 1.0;
#if (H100 != 1)
#if USE_FP64 
  hipsparseDcsrsv2_solve(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			n, 
			nnzU, 
			&one, 
			descrU,
			ua,
			uia,
			uja,
			infoU,
			x,
			result,
			policy,
			U_buffer);
#else
  hipsparseScsrsv2_solve(handle_cusparse, 
			HIPSPARSE_OPERATION_NON_TRANSPOSE, 
			n, 
			nnzU, 
			&one, 
			descrU,
			ua,
			uia,
			uja,
			infoU,
			x,
			result,
			policy,
			U_buffer);
#endif
#else // cuda 12 or H100
  hipsparseDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseCreateDnVec(&vecX, n, (double *) x, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, n, result, HIP_R_64F);
  hipsparseSpSV_solve(handle_cusparse, 
		     HIPSPARSE_OPERATION_NON_TRANSPOSE,
		     &one, 
		     matU, 
		     vecX, 
		     vecY, 
		     HIP_R_64F,
		     HIPSPARSE_SPSV_ALG_DEFAULT, 
		     descrU); 
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);


#endif
}

/* not std blas but needed and embarassingly parallel */

/* cuda vec-vec computes an element-wise product (needed for scaling) */

void cuda_vec_vec(const int n, const real_type *x, const real_type *y, real_type *res){

  cuda_vec_vec_kernel<<<1024, 1024>>>(n, x, y, res);
}

/* vector reciprocal computes 1./d */ 

void cuda_vector_reciprocal(const int n, const real_type *v, real_type *res){

  cuda_vec_reciprocal_kernel<<<1024, 1024>>>(n, v, res);
}

/* vector sqrt takes an sqrt from each vector entry */

void cuda_vector_sqrt(const int n, const real_type *v, real_type *res){

  cuda_vec_sqrt_kernel<<<1024, 1024>>>(n, v, res);
}

void cuda_vec_copy(const int n, const real_type *src, real_type *dest){

  hipMemcpy(dest, src, sizeof(real_type) * n, hipMemcpyDeviceToDevice);
}

void cuda_vec_zero(const int n, real_type *vec){

  cuda_vec_zero_kernel<<<1024, 1024>>>(n, vec);
}

